#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

typedef struct Node {
    int r_len = 0;
    int dest[10];
    float len[10];
} Node;

Node road[10000000];
float lan[10000000];
float lot[10000000];


vector<double> arr;
vector<float> tmp;

__global__ void Print(Node *a){
    printf("%d\n", a[999].dest[0]);
}

__device__ Node *gpu_road;
__device__ int start_node, end_node;

void GetTmp(string str) {
    string s;

    for (int i = 0; i < str.size(); ++i) {
        if (str[i] != ',')
            s.push_back(str[i]);
        else {
            tmp.push_back(stof(s));
            s.clear();
        }
    }
    tmp.push_back(stof(s));
}

int main(void)
{
    ifstream node, edge;

    string filename("result_noded.csv");

    node.open("result_noded.csv", ios::in);

    vector<string> s;
    if (node.is_open())
    {
        string str;
        getline(node, str);

        int i = 0;
        while (!node.eof())
        {
            printf("%d\n", i++);
            string delim = ",";

            std::getline(node, str);
            if (str == "")
                break;
            GetTmp(str);

            lan[(int)tmp[1]] = tmp[6];
            lot[(int)tmp[1]] = tmp[7];

            if ((int)tmp[4]!= 0) {
                road[(int)tmp[1]].dest[road[(int)tmp[1]].r_len] = (int)tmp[2];
                road[(int)tmp[1]].len[road[(int)tmp[1]].r_len++] = tmp[3];
            }
            if ((int)tmp[5] != 0) {
                road[(int)tmp[2]].dest[road[(int)tmp[2]].r_len] = (int)tmp[1];
                road[(int)tmp[2]].len[road[(int)tmp[2]].r_len++] = tmp[3];
            }
            tmp.clear();
        }
        edge.close();
    }

    hipMalloc(&gpu_road, 10000000 * sizeof(Node));
    hipMemcpy(gpu_road, road, 10000000 * sizeof(Node), hipMemcpyHostToDevice);

    Print<<<1,1>>>(gpu_road);
    hipDeviceSynchronize();

    return 0;
}
