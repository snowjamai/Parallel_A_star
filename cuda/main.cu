#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>

#include "Route.cuh"

using namespace std;

typedef struct Node {
    int r_len = 0;
    int dest[10];
    float len[10];
} Node;

Node road[10000000];
float lat[10000000];
float lon[10000000];



vector<double> arr;
vector<float> tmp;


__device__ Node *gpu_road;
__device__ int *start_node, *end_node, *start_len, *end_len;
__device__ float *gpu_lat, *gpu_lon;

__global__ void Print(int *a, int *node){
    for(int i = 0; i < *a; ++i)
        printf("***%d\n", node[i]);
}

void GetTmp(string str) {
    string s;

    for (int i = 0; i < str.size(); ++i) {
        if (str[i] != ',')
            s.push_back(str[i]);
        else {
            tmp.push_back(stof(s));
            s.clear();
        }
    }
    tmp.push_back(stof(s));
}

int main(void)
{
    ifstream node, edge;

    string filename("result_noded.csv");

    node.open("result_noded.csv", ios::in);

    vector<string> s;
    if (node.is_open())
    {
        string str;
        getline(node, str);

        int i = 0;
        while (!node.eof())
        {
            printf("%d\n", i++);
            string delim = ",";

            std::getline(node, str);
            if (str == "")
                break;
            GetTmp(str);

            lat[(int)tmp[1]] = tmp[6];
            lon[(int)tmp[1]] = tmp[7];

            if ((int)tmp[4]!= 0) {
                road[(int)tmp[1]].dest[road[(int)tmp[1]].r_len] = (int)tmp[2];
                road[(int)tmp[1]].len[road[(int)tmp[1]].r_len++] = tmp[3];
            }
            if ((int)tmp[5] != 0) {
                road[(int)tmp[2]].dest[road[(int)tmp[2]].r_len] = (int)tmp[1];
                road[(int)tmp[2]].len[road[(int)tmp[2]].r_len++] = tmp[3];
            }
            tmp.clear();
        }
        edge.close();
    }

    hipMalloc(&gpu_road, 10000000 * sizeof(Node));
    hipMemcpy(gpu_road, road, 10000000 * sizeof(Node), hipMemcpyHostToDevice);

    hipMalloc(&gpu_lat, 10000000 * sizeof(float));
    hipMemcpy(gpu_lat, lat, 10000000 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&gpu_lon, 10000000 * sizeof(float));
    hipMemcpy(gpu_lon, lon, 10000000 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&start_len, sizeof(int));
    hipMalloc(&start_node, sizeof(int) * 10);
    hipMalloc(&end_len, sizeof(int));
    hipMalloc(&end_node, sizeof(int) * 10);


    FindStart<<<1,1>>>(gpu_lat,gpu_lon, 126.972778,37.556328, start_node, start_len);
    FindEnd<<<1,1>>>(gpu_lat,gpu_lon, 129.042049,35.115294, end_node, end_len);
    hipDeviceSynchronize();




    Print<<<1,1>>>(start_len, start_node);
    Print<<<1,1>>>(end_len, end_node);
    hipDeviceSynchronize();

    return 0;
}
