#include "hip/hip_runtime.h"
//
// Created by snowjam02 on 5/25/21.
//
#include <iostream>
#include "Route.cuh"
__device__ float epsilon = 0.001;

__global__ void FindStart(float *gpu_lat,float *gpu_lon, float lat, float lon, int *start, int *start_len){
    printf("%d\n", *start_len);
    for(int i = 1; i < 10000000; ++i) {
        if (gpu_lat[i] >= lat - epsilon && gpu_lat[i] < lat + epsilon && gpu_lon[i] >= lon - epsilon && gpu_lon[i] < lon + epsilon)
            start[(*start_len)++] = i;
    }
    printf("%d\n", *start_len);
    return;
}
__global__ void FindEnd(float *gpu_lat, float *gpu_lon, float lat, float lon, int *end, int *end_len) {
    for(int i = 1; i < 10000000; ++i)
        if(gpu_lat[i] >= lat - epsilon && gpu_lat[i] < lat + epsilon && gpu_lon[i] >= lon - epsilon && gpu_lon[i] < lon + epsilon)
            end[(*end_len)++] = i;
    printf("%d\n", *end_len);
    return;
}